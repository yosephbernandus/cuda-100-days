
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstdlib>

__global__ void vecAddKernel(float *A, float *B, float *C, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < n) {
    C[i] = A[i] + B[i];
  }
}

void vecAdd(float *A, float *B, float *C, int n) {
  float *A_d, *B_d, *C_d;
  int size = n * sizeof(float);

  hipMalloc((void **)&A_d, size);
  hipMalloc((void **)&B_d, size);
  hipMalloc((void **)&C_d, size);

  hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);

  vecAddKernel<<<ceil(n / 256.0), 256>>>(A_d, B_d, C_d, n);

  hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
}

int main() {
  int n = 128;
  size_t size = n * sizeof(float);
  float *A_h = (float *)malloc(size);
  float *B_h = (float *)malloc(size);
  float *C_h = (float *)malloc(size);

  for (size_t i = 0; i < n; i++) {
    A_h[i] = 1;
    B_h[i] = 2;
  }

  vecAdd(A_h, B_h, C_h, n);

  for (size_t i = 0; i < n; i++) {
    assert(C_h[i] == 3);
  }

  return 0;
}
